#include "hip/hip_runtime.h"
/*
 -*- coding: utf-8 -*-
Max-Planck-Gesellschaft zur Förderung der Wissenschaften e.V. (MPG) is
holder of all proprietary rights on this computer program.
You can only use this computer program if you have closed
a license agreement with MPG or you get the right to use the computer
program from someone who is authorized to grant you that right.
Any use of the computer program without a valid license is prohibited and
liable to prosecution.

Copyright©2023 Max-Planck-Gesellschaft zur Förderung
der Wissenschaften e.V. (MPG). acting on behalf of its Max Planck Institute
for Intelligent Systems. All rights reserved.

Contact: insta@tue.mpg.de
*/

#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/common.h>

#include <rta/core.h>

#include <args/args.hxx>

#include <filesystem/path.h>

// billboards
#include <thread>
#include <websocketpp/config/asio_no_tls.hpp>
#include <websocketpp/server.hpp>
#include <sstream>
#include <vector>
#include <functional>

using namespace args;
using namespace ngp;
using namespace std;
using namespace tcnn;
namespace fs = ::filesystem;


// billboards
typedef websocketpp::server<websocketpp::config::asio> server;
rta::Core core(ETestbedMode::Nerf);
server s;

Eigen::Matrix<float, 3, 4> convertToMatrix(const std::string& input) {
    std::stringstream ss(input);
    float value;
    std::vector<float> values;
    while (ss >> value) {
        values.push_back(value);
        if (ss.peek() == ',')
            ss.ignore();
    }
    Eigen::Matrix<float, 3, 4> result;
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 4; ++j) {
            result(i, j) = values[j * 4 + i];
        }
    }
    return result;
}

void on_message(websocketpp::connection_hdl hdl, server::message_ptr msg) {
    Eigen::Matrix<float, 3, 4> mat = convertToMatrix(msg->get_payload());

    // ref: rta::Recorder::video()
    core.m_recorder->m_ngp->m_target_deform_frame = core.m_recorder->m_index_frame;
    core.m_recorder->m_ngp->m_nerf.extra_dim_idx_for_inference = core.m_recorder->m_index_frame;
    core.m_recorder->m_dst_folder = "floating";
    core.m_recorder->m_to_record = core.m_nerf.training.dataset.n_all_images;

    // ref: rta::Recorder::start()
    core.m_dynamic_res = false;

    core.m_camera = mat;
    core.m_zoom = 5.f;
    std::vector<uint8_t> pngpixels = core.m_recorder->dump_frame_buffer();
    try {
        s.send(hdl, pngpixels.data(), pngpixels.size(), websocketpp::frame::opcode::binary);
    } catch (websocketpp::exception const & e) {
        std::cout << "Error sending message: " << e.what() << std::endl;
    }
}

void start_server(server& s) {
    s.clear_access_channels(websocketpp::log::alevel::all);
    s.clear_error_channels(websocketpp::log::elevel::all);
    s.set_message_handler(&on_message);
    s.init_asio();
    s.listen(9002);
    s.start_accept();
    s.run();
}

int main(int argc, char **argv) {
    ArgumentParser parser{
            "neural graphics primitives\n"
            "version " NGP_VERSION,
            "",
    };

    HelpFlag help_flag{
            parser,
            "HELP",
            "Display this help menu.",
            {'h', "help"},
    };

    ValueFlag<string> mode_flag{
            parser,
            "MODE",
            "Mode can be 'nerf', 'sdf', or 'image' or 'volume'. Inferred from the scene if unspecified.",
            {'m', "mode"},
    };

    ValueFlag<string> network_config_flag{
            parser,
            "CONFIG",
            "Path to the network config. Uses the scene's default if unspecified.",
            {'n', 'c', "network", "config"},
    };

    Flag no_gui_flag{
            parser,
            "NO_GUI",
            "Disables the GUI and instead reports training progress on the command line.",
            {"no-gui"},
    };

    Flag no_train_flag{
            parser,
            "NO_TRAIN",
            "Disables training on startup.",
            {"no-train"},
    };

    ValueFlag<string> scene_flag{
            parser,
            "SCENE",
            "The scene to load. Can be NeRF dataset, a *.obj mesh for training a SDF, an image, or a *.nvdb volume.",
            {'s', "scene"},
    };

    ValueFlag<string> snapshot_flag{
            parser,
            "SNAPSHOT",
            "Optional snapshot to load upon startup.",
            {"snapshot"},
    };

    ValueFlag<uint32_t> width_flag{
            parser,
            "WIDTH",
            "Resolution width of the GUI.",
            {"width"},
    };

    ValueFlag<uint32_t> height_flag{
            parser,
            "HEIGHT",
            "Resolution height of the GUI.",
            {"height"},
    };

    Flag version_flag{
            parser,
            "VERSION",
            "Display the version of neural graphics primitives.",
            {'v', "version"},
    };

    // Parse command line arguments and react to parsing
    // errors using exceptions.
    try {
        parser.ParseCLI(argc, argv);
    } catch (const Help &) {
        cout << parser;
        return 0;
    } catch (const ParseError &e) {
        cerr << e.what() << endl;
        cerr << parser;
        return -1;
    } catch (const ValidationError &e) {
        cerr << e.what() << endl;
        cerr << parser;
        return -2;
    }

    if (version_flag) {
        tlog::none() << "neural graphics primitives version " NGP_VERSION;
        return 0;
    }

    try {
        ETestbedMode mode = ETestbedMode::Nerf;
        if (!mode_flag) {
            if (!scene_flag) {
                tlog::error() << "Must specify either a mode or a scene";
                return 1;
            }

            fs::path scene_path = get(scene_flag);
            if (!scene_path.exists()) {
                tlog::error() << "Scene path " << scene_path << " does not exist.";
                return 1;
            }

            if (scene_path.is_directory() || equals_case_insensitive(scene_path.extension(), "json")) {
                mode = ETestbedMode::Nerf;
            } else if (equals_case_insensitive(scene_path.extension(), "obj") || equals_case_insensitive(scene_path.extension(), "stl")) {
                mode = ETestbedMode::Sdf;
            } else if (equals_case_insensitive(scene_path.extension(), "nvdb")) {
                mode = ETestbedMode::Volume;
            } else {
                mode = ETestbedMode::Image;
            }
        } else {
            auto mode_str = get(mode_flag);
            if (equals_case_insensitive(mode_str, "nerf")) {
                mode = ETestbedMode::Nerf;
            } else if (equals_case_insensitive(mode_str, "sdf")) {
                mode = ETestbedMode::Sdf;
            } else if (equals_case_insensitive(mode_str, "image")) {
                mode = ETestbedMode::Image;
            } else if (equals_case_insensitive(mode_str, "volume")) {
                mode = ETestbedMode::Volume;
            } else {
                tlog::error() << "Mode must be one of 'nerf', 'sdf', 'image', and 'volume'.";
                return 1;
            }
        }

        // billboards
        // rta::Core core(mode);

        std::string mode_str;
        switch (mode) {
            case ETestbedMode::Nerf:
                mode_str = "nerf";
                break;
            case ETestbedMode::Sdf:
                mode_str = "sdf";
                break;
            case ETestbedMode::Image:
                mode_str = "image";
                break;
            case ETestbedMode::Volume:
                mode_str = "volume";
                break;
        }

        // Otherwise, load the network config and prepare for training
        fs::path network_config_path = fs::path{"configs"} / mode_str;
        if (network_config_flag) {
            auto network_config_str = get(network_config_flag);
            if ((network_config_path / network_config_str).exists()) {
                network_config_path = network_config_path / network_config_str;
            } else {
                network_config_path = network_config_str;
            }
        } else {
            network_config_path = network_config_path / "base.json";
        }

        if (!network_config_path.exists()) {
            tlog::error() << "Network config path " << network_config_path << " does not exist.";
            return 1;
        }

        core.reload_network_from_file(network_config_path.str());

        if (scene_flag) {
            fs::path scene_path = get(scene_flag);
            if (!scene_path.exists()) {
                tlog::error() << "Scene path " << scene_path << " does not exist.";
                return 1;
            }
            core.load_training_data(scene_path.str());
        }

        core.m_train = !no_train_flag;
        core.post_loading();

        if (snapshot_flag) {
            // Load network from a snapshot if one is provided
            fs::path snapshot_path = get(snapshot_flag);
            if (!snapshot_path.exists()) {
                tlog::error() << "Snapshot path " << snapshot_path << " does not exist.";
                return 1;
            }

            core.load_snapshot(snapshot_path.str());

            core.m_train = false;
            core.m_offscreen_rendering = false;
        }

        bool gui = !no_gui_flag;
#ifndef NGP_GUI
        gui = false;
#endif
        // auto W = width_flag ? get(width_flag) : 1024;
        // auto H = height_flag ? get(height_flag) : 1024;
        auto W = 800;
        auto H = 800;

        if (gui) {
            core.init_window(W, H);
        } else {
            core.init_render_surface(W, H);
        }

        core.is_using_gui = gui;

        // billboards
        std::thread server_thread(start_server, std::ref(s));

        // Render/training loop
        while (core.frame()) {
            if (!gui) {
                if (core.is_recording())
                    tlog::info() << core.get_recorder_info();
                else
                    tlog::info() << "iteration=" << core.m_training_step << " loss=" << core.m_loss_scalar.val();
            }
        }
    } catch (const exception &e) {
        tlog::error() << "Uncaught exception: " << e.what();
        return 1;
    }
}
