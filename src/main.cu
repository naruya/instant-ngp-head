/*
 -*- coding: utf-8 -*-
Max-Planck-Gesellschaft zur Förderung der Wissenschaften e.V. (MPG) is
holder of all proprietary rights on this computer program.
You can only use this computer program if you have closed
a license agreement with MPG or you get the right to use the computer
program from someone who is authorized to grant you that right.
Any use of the computer program without a valid license is prohibited and
liable to prosecution.

Copyright©2023 Max-Planck-Gesellschaft zur Förderung
der Wissenschaften e.V. (MPG). acting on behalf of its Max Planck Institute
for Intelligent Systems. All rights reserved.

Contact: insta@tue.mpg.de
*/

#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/common.h>

#include <rta/core.h>

#include <args/args.hxx>

#include <filesystem/path.h>

// billboards
#include <thread>
#include <websocketpp/config/asio_no_tls.hpp>
#include <websocketpp/server.hpp>
#include <sstream>
#include <vector>

using namespace args;
using namespace ngp;
using namespace std;
using namespace tcnn;
namespace fs = ::filesystem;


// billboards
typedef websocketpp::server<websocketpp::config::asio> server;
ETestbedMode mode = ETestbedMode::Nerf;
rta::Core core(mode);

Eigen::Matrix<float, 3, 4> convertToMatrix(const std::string& input) {
    std::stringstream ss(input);
    float value;
    std::vector<float> values;
    while (ss >> value) {
        values.push_back(value);
        if (ss.peek() == ',')
            ss.ignore();
    }
    Eigen::Matrix<float, 3, 4> result;
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 4; ++j) {
            result(i, j) = values[j * 4 + i];
        }
    }
    return result;
}

void on_message(websocketpp::connection_hdl hdl, server::message_ptr msg) {
    Eigen::Matrix<float, 3, 4> mat = convertToMatrix(msg->get_payload());

    core.m_camera = mat;
    core.m_recorder->m_video_mode = rta::VideoType::Floating;
    core.m_recorder->dump_frame_buffer();
    // std::cout << msg->get_payload() << std::endl;
    // std::cout << core.m_camera << std::endl;
}

void start_server() {
    server print_server;
    print_server.set_message_handler(&on_message);
    print_server.init_asio();
    print_server.listen(9003);
    print_server.start_accept();
    print_server.run();
}


int main(int argc, char **argv) {
    ArgumentParser parser{
            "neural graphics primitives\n"
            "version " NGP_VERSION,
            "",
    };

    HelpFlag help_flag{
            parser,
            "HELP",
            "Display this help menu.",
            {'h', "help"},
    };

    ValueFlag<string> mode_flag{
            parser,
            "MODE",
            "Mode can be 'nerf', 'sdf', or 'image' or 'volume'. Inferred from the scene if unspecified.",
            {'m', "mode"},
    };

    ValueFlag<string> network_config_flag{
            parser,
            "CONFIG",
            "Path to the network config. Uses the scene's default if unspecified.",
            {'n', 'c', "network", "config"},
    };

    Flag no_gui_flag{
            parser,
            "NO_GUI",
            "Disables the GUI and instead reports training progress on the command line.",
            {"no-gui"},
    };

    Flag no_train_flag{
            parser,
            "NO_TRAIN",
            "Disables training on startup.",
            {"no-train"},
    };

    ValueFlag<string> scene_flag{
            parser,
            "SCENE",
            "The scene to load. Can be NeRF dataset, a *.obj mesh for training a SDF, an image, or a *.nvdb volume.",
            {'s', "scene"},
    };

    ValueFlag<string> snapshot_flag{
            parser,
            "SNAPSHOT",
            "Optional snapshot to load upon startup.",
            {"snapshot"},
    };

    ValueFlag<uint32_t> width_flag{
            parser,
            "WIDTH",
            "Resolution width of the GUI.",
            {"width"},
    };

    ValueFlag<uint32_t> height_flag{
            parser,
            "HEIGHT",
            "Resolution height of the GUI.",
            {"height"},
    };

    Flag version_flag{
            parser,
            "VERSION",
            "Display the version of neural graphics primitives.",
            {'v', "version"},
    };

    // Parse command line arguments and react to parsing
    // errors using exceptions.
    try {
        parser.ParseCLI(argc, argv);
    } catch (const Help &) {
        cout << parser;
        return 0;
    } catch (const ParseError &e) {
        cerr << e.what() << endl;
        cerr << parser;
        return -1;
    } catch (const ValidationError &e) {
        cerr << e.what() << endl;
        cerr << parser;
        return -2;
    }

    if (version_flag) {
        tlog::none() << "neural graphics primitives version " NGP_VERSION;
        return 0;
    }

    try {
        ETestbedMode mode = ETestbedMode::Nerf;
        if (!mode_flag) {
            if (!scene_flag) {
                tlog::error() << "Must specify either a mode or a scene";
                return 1;
            }

            fs::path scene_path = get(scene_flag);
            if (!scene_path.exists()) {
                tlog::error() << "Scene path " << scene_path << " does not exist.";
                return 1;
            }

            if (scene_path.is_directory() || equals_case_insensitive(scene_path.extension(), "json")) {
                mode = ETestbedMode::Nerf;
            } else if (equals_case_insensitive(scene_path.extension(), "obj") || equals_case_insensitive(scene_path.extension(), "stl")) {
                mode = ETestbedMode::Sdf;
            } else if (equals_case_insensitive(scene_path.extension(), "nvdb")) {
                mode = ETestbedMode::Volume;
            } else {
                mode = ETestbedMode::Image;
            }
        } else {
            auto mode_str = get(mode_flag);
            if (equals_case_insensitive(mode_str, "nerf")) {
                mode = ETestbedMode::Nerf;
            } else if (equals_case_insensitive(mode_str, "sdf")) {
                mode = ETestbedMode::Sdf;
            } else if (equals_case_insensitive(mode_str, "image")) {
                mode = ETestbedMode::Image;
            } else if (equals_case_insensitive(mode_str, "volume")) {
                mode = ETestbedMode::Volume;
            } else {
                tlog::error() << "Mode must be one of 'nerf', 'sdf', 'image', and 'volume'.";
                return 1;
            }
        }

        // billboards
        // rta::Core core(mode);

        std::string mode_str;
        switch (mode) {
            case ETestbedMode::Nerf:
                mode_str = "nerf";
                break;
            case ETestbedMode::Sdf:
                mode_str = "sdf";
                break;
            case ETestbedMode::Image:
                mode_str = "image";
                break;
            case ETestbedMode::Volume:
                mode_str = "volume";
                break;
        }

        // Otherwise, load the network config and prepare for training
        fs::path network_config_path = fs::path{"configs"} / mode_str;
        if (network_config_flag) {
            auto network_config_str = get(network_config_flag);
            if ((network_config_path / network_config_str).exists()) {
                network_config_path = network_config_path / network_config_str;
            } else {
                network_config_path = network_config_str;
            }
        } else {
            network_config_path = network_config_path / "base.json";
        }

        if (!network_config_path.exists()) {
            tlog::error() << "Network config path " << network_config_path << " does not exist.";
            return 1;
        }

        core.reload_network_from_file(network_config_path.str());

        if (scene_flag) {
            fs::path scene_path = get(scene_flag);
            if (!scene_path.exists()) {
                tlog::error() << "Scene path " << scene_path << " does not exist.";
                return 1;
            }
            core.load_training_data(scene_path.str());
        }

        core.m_train = !no_train_flag;
        core.post_loading();

        if (snapshot_flag) {
            // Load network from a snapshot if one is provided
            fs::path snapshot_path = get(snapshot_flag);
            if (!snapshot_path.exists()) {
                tlog::error() << "Snapshot path " << snapshot_path << " does not exist.";
                return 1;
            }

            core.load_snapshot(snapshot_path.str());

            core.m_train = false;
            core.m_offscreen_rendering = false;
        }

        bool gui = !no_gui_flag;
#ifndef NGP_GUI
        gui = false;
#endif
        auto W = width_flag ? get(width_flag) : 1024;
        auto H = height_flag ? get(height_flag) : 1024;

        if (gui) {
            core.init_window(W, H);
        } else {
            core.init_render_surface(W, H);
        }

        core.is_using_gui = gui;

        // billboards
        std::thread server_thread(start_server);

        // Render/training loop
        while (core.frame()) {
            if (!gui) {
                if (core.is_recording())
                    tlog::info() << core.get_recorder_info();
                else
                    tlog::info() << "iteration=" << core.m_training_step << " loss=" << core.m_loss_scalar.val();
            }
        }
    } catch (const exception &e) {
        tlog::error() << "Uncaught exception: " << e.what();
        return 1;
    }
}
